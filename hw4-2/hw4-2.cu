#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <omp.h>

#define SIZEOFINT sizeof(int)
#define BLOCK_DIM 24
#define TH_DIM 24

const int INF = ((1 << 30) - 1);
int n, m, padding_n;
int *Dist, *Dist_s;
// int *Dist_cuda;
int up_part_b_size = 0, bottom_part_b_size = 0;
int *Dist_cuda0, *Dist_cuda1;

void show_mat(int *start_p, int vertex_num){
    for(int i = 0; i < vertex_num; i++){
        for(int j = 0; j < vertex_num; j++){
            if(start_p[i * vertex_num + j] == INF){
                printf("INF\t  ");
            }else{
                printf("%d\t  ", start_p[i * vertex_num + j]);
            }   
        }
        printf("\n");
    }
}

void show_mat_cuda(int *start_p, int vertex_num, int device_id){
    int *temp = (int*)malloc(SIZEOFINT * vertex_num * vertex_num);
    hipSetDevice(device_id);
    hipMemcpy(temp, start_p, (SIZEOFINT * vertex_num * vertex_num), hipMemcpyDeviceToHost);

    for(int i = 0; i < vertex_num; i++){
        for(int j = 0; j < vertex_num; j++){
            if(temp[i * vertex_num + j] == INF){
                printf("INF\t  ");
            }else{
                printf("%d\t  ", temp[i * vertex_num + j]);
            }   
        }
        printf("\n");
    }
}

void malloc_Dist(){
    hipHostAlloc(&Dist, SIZEOFINT * padding_n * padding_n, hipHostMallocPortable);
    // Dist = (int*)malloc(SIZEOFINT * padding_n * padding_n);
    Dist_s = (int*)malloc(SIZEOFINT * n * n);
}
int getDist(int i, int j){return Dist[i * padding_n + j];}
int *getDistAddr(int i, int j){return &(Dist[i * padding_n + j]);}
void setDist(int i, int j, int val){Dist[i * padding_n + j] = val;}

void setup_DistCuda(){
    // cudaMalloc((void **)&Dist_cuda, SIZEOFINT * padding_n * padding_n);
    // cudaMemcpy(Dist_cuda, Dist, (padding_n * padding_n * SIZEOFINT), cudaMemcpyHostToDevice);

    // int *Dist_cudas[2];
    // Dist_cudas[0] = Dist_cuda0;
    // Dist_cudas[1] = Dist_cuda1;
    // #pragma omp parallel
    // for(int i=0; i<2; i++){
    //     cudaSetDevice(i);
    //     cudaDeviceEnablePeerAccess(i, 0);
    //     cudaMalloc((void **)&(Dist_cudas[i]), SIZEOFINT * padding_n * padding_n);
    //     cudaMemcpy((Dist_cudas[i]), Dist, (padding_n * padding_n * SIZEOFINT), cudaMemcpyHostToDevice);
    // }

    hipStream_t stream;
    hipStreamCreate(&stream);

    hipSetDevice(0);
    hipDeviceEnablePeerAccess(0, 0);
    hipMalloc((void **)&Dist_cuda0, SIZEOFINT * padding_n * padding_n);
    hipMemcpyAsync(Dist_cuda0, Dist, (padding_n * padding_n * SIZEOFINT), hipMemcpyHostToDevice, stream);

    hipSetDevice(1);
    hipDeviceEnablePeerAccess(1, 0);
    hipMalloc((void **)&Dist_cuda1, SIZEOFINT * padding_n * padding_n);
    hipMemcpy(Dist_cuda1, Dist, (padding_n * padding_n * SIZEOFINT), hipMemcpyHostToDevice);

    hipStreamDestroy(stream);
}
void back_DistCuda(){
    // cudaMemcpy(Dist, Dist_cuda, (padding_n * padding_n * SIZEOFINT), cudaMemcpyDeviceToHost);

    hipStream_t stream;
    hipStreamCreate(&stream);

    hipSetDevice(0);
    hipMemcpyAsync(Dist, Dist_cuda0, (BLOCK_DIM * up_part_b_size * padding_n * SIZEOFINT), hipMemcpyDeviceToHost, stream);

    hipSetDevice(1);
    hipMemcpy(&(Dist[BLOCK_DIM * up_part_b_size * padding_n]), &(Dist_cuda1[BLOCK_DIM * up_part_b_size * padding_n]), (BLOCK_DIM * bottom_part_b_size * padding_n * SIZEOFINT), hipMemcpyDeviceToHost);

    hipStreamDestroy(stream);
}

void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);
    padding_n = ((n + BLOCK_DIM - 1) / BLOCK_DIM) * BLOCK_DIM;
    malloc_Dist();

    for (int i = 0; i < padding_n; i++) {
        for (int j = 0; j < padding_n; j++) {
            if (i == j) {
                setDist(i, j, 0);
                // Dist[i][j] = 0;
            } else {
                setDist(i, j, INF);
                // Dist[i][j] = INF;
            }
        }
    }

    int pair[3];
    for (int i = 0; i < m; i++) {
        fread(pair, sizeof(int), 3, file);
        setDist(pair[0], pair[1], pair[2]);
        // Dist[pair[0]][pair[1]] = pair[2];
    }
    // cudaMemcpy(Dist_cuda, Dist, (n * n * SIZEOFINT), cudaMemcpyHostToDevice);
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            // if (Dist[i][j] >= INF) Dist[i][j] = INF;
            if (getDist(i, j) >= INF) setDist(i, j, INF);
            Dist_s[i * n + j] = getDist(i, j);
        }
        // fwrite(Dist[i], sizeof(int), n, outfile);
        // fwrite(getDistAddr(i, 0), SIZEOFINT, n, outfile);
    }
    fwrite(Dist_s, sizeof(int), n * n, outfile);
    fclose(outfile);
}

__forceinline__
__device__ void block_calc(int* C, int* A, int* B, int bj, int bi) {
    for (int k = 0; k < BLOCK_DIM; k++) {
        int sum0 = A[bi*BLOCK_DIM + k] + B[k*BLOCK_DIM + bj];
        // int sum1 = A[(bi + TH_DIM)*BLOCK_DIM + k] + B[k*BLOCK_DIM + bj];
        // int sum2 = A[bi*BLOCK_DIM + k] + B[k*BLOCK_DIM + (bj + TH_DIM)];
        // int sum3 = A[(bi + TH_DIM)*BLOCK_DIM + k] + B[k*BLOCK_DIM + (bj + TH_DIM)];

        C[bi*BLOCK_DIM + bj] = min(C[bi*BLOCK_DIM + bj], sum0);
        // C[(bi + TH_DIM)*BLOCK_DIM + bj] = min(C[(bi + TH_DIM)*BLOCK_DIM + bj], sum1);
        // C[bi*BLOCK_DIM + (bj + TH_DIM)] = min(C[bi*BLOCK_DIM + (bj + TH_DIM)], sum2);
        // C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = min(C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)], sum3);
        __syncthreads();
    }
}

__forceinline__
__device__ void block_calc_rev_async(int* C, int* A, int* B, int bj, int bi) {
    #pragma unroll 10
    for (int k = 0; k < BLOCK_DIM; k++) {
        int sum0 = A[k*BLOCK_DIM + bi] + B[k*BLOCK_DIM + bj];
        // int sum1 = A[k*BLOCK_DIM + (bi + TH_DIM)] + B[k*BLOCK_DIM + bj];
        // int sum2 = A[k*BLOCK_DIM + bi] + B[k*BLOCK_DIM + (bj + TH_DIM)];
        // int sum3 = A[k*BLOCK_DIM + (bi + TH_DIM)] + B[k*BLOCK_DIM + (bj + TH_DIM)];

        C[bi*BLOCK_DIM + bj] = min(C[bi*BLOCK_DIM + bj], sum0);
        // C[(bi + TH_DIM)*BLOCK_DIM + bj] = min(C[(bi + TH_DIM)*BLOCK_DIM + bj], sum1);
        // C[bi*BLOCK_DIM + (bj + TH_DIM)] = min(C[bi*BLOCK_DIM + (bj + TH_DIM)], sum2);
        // C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = min(C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)], sum3);
    }
}

__global__ void floyd_warshall_block_kernel_phase1_mw(int n, int k, int* graph, int *dst0, int *dst1) {
    const unsigned int bi = threadIdx.y;
    const unsigned int bj = threadIdx.x;

    __shared__ int C[BLOCK_DIM * BLOCK_DIM];

    // Transfer to temp shared arrays
    C[bi*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + bj)];
    // C[(bi + TH_DIM)*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + bj)];
    // C[bi*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + (bj + TH_DIM))];
    // C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + (bj + TH_DIM))];

    __syncthreads();
    
    block_calc(C, C, C, bi, bj);

    // Transfer back to graph
    dst0[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + bj)] = C[bi*BLOCK_DIM + bj];
    // dst0[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + bj)] = C[(bi + TH_DIM)*BLOCK_DIM + bj];
    // dst0[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + (bj + TH_DIM))] = C[bi*BLOCK_DIM + (bj + TH_DIM)];
    // dst0[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + (bj + TH_DIM))] = C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)];

    dst1[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + bj)] = C[bi*BLOCK_DIM + bj];
    // dst1[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + bj)] = C[(bi + TH_DIM)*BLOCK_DIM + bj];
    // dst1[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + (bj + TH_DIM))] = C[bi*BLOCK_DIM + (bj + TH_DIM)];
    // dst1[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + (bj + TH_DIM))] = C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)];
}

__global__ void floyd_warshall_block_kernel_phase1(int n, int k, int* graph) {
    const unsigned int bi = threadIdx.y;
    const unsigned int bj = threadIdx.x;

    __shared__ int C[BLOCK_DIM * BLOCK_DIM];

    // Transfer to temp shared arrays
    C[bi*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + bj)];
    // C[(bi + TH_DIM)*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + bj)];
    // C[bi*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + (bj + TH_DIM))];
    // C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + (bj + TH_DIM))];

    __syncthreads();
    
    block_calc(C, C, C, bi, bj);

    // Transfer back to graph
    graph[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + bj)] = C[bi*BLOCK_DIM + bj];
    // graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + bj)] = C[(bi + TH_DIM)*BLOCK_DIM + bj];
    // graph[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + (bj + TH_DIM))] = C[bi*BLOCK_DIM + (bj + TH_DIM)];
    // graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + (bj + TH_DIM))] = C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)];
}


__global__ void floyd_warshall_block_kernel_phase2(int n, int k, int* graph) {
    // BlockDim is one dimensional (Straight along diagonal)
    // Blocks themselves are two dimensional
    // Phase 2 1/2
    const unsigned int i = blockIdx.x;
    const unsigned int bi = threadIdx.y;
    const unsigned int bj = threadIdx.x;

    __shared__ int A[BLOCK_DIM * BLOCK_DIM];
    __shared__ int B[BLOCK_DIM * BLOCK_DIM];
    __shared__ int C[BLOCK_DIM * BLOCK_DIM];

    C[bi*BLOCK_DIM + bj] = graph[(i*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + bj)];
    // C[(bi + TH_DIM)*BLOCK_DIM + bj] = graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + bj)];
    // C[bi*BLOCK_DIM + (bj + TH_DIM)] = graph[(i*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + (bj + TH_DIM))];
    // C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + (bj + TH_DIM))];

    B[bi*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + bj)];
    // B[(bi + TH_DIM)*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + bj)];
    // B[bi*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + (bj + TH_DIM))];
    // B[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + (bj + TH_DIM))];

    __syncthreads();

    block_calc(C, C, B, bi, bj);

    graph[(i*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + bj)] = C[bi*BLOCK_DIM + bj];
    // graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + bj)] = C[(bi + TH_DIM)*BLOCK_DIM + bj];
    // graph[(i*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + (bj + TH_DIM))] = C[bi*BLOCK_DIM + (bj + TH_DIM)];
    // graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + (bj + TH_DIM))] = C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)];

    // Phase 2 2/2

    C[bi*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + bi)*n + (i*BLOCK_DIM + bj)];
    // C[(bi + TH_DIM)*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (i*BLOCK_DIM + bj)];
    // C[bi*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + bi)*n + (i*BLOCK_DIM + (bj + TH_DIM))];
    // C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (i*BLOCK_DIM + (bj + TH_DIM))];

    __syncthreads();

    block_calc(C, B, C, bi, bj);

    // Block C is the only one that could be changed
    graph[(k*BLOCK_DIM + bi)*n + (i*BLOCK_DIM + bj)] = C[bi*BLOCK_DIM + bj];
    // graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (i*BLOCK_DIM + bj)] = C[(bi + TH_DIM)*BLOCK_DIM + bj];
    // graph[(k*BLOCK_DIM + bi)*n + (i*BLOCK_DIM + (bj + TH_DIM))] = C[bi*BLOCK_DIM + (bj + TH_DIM)];
    // graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (i*BLOCK_DIM + (bj + TH_DIM))] = C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)];
}


__global__ void floyd_warshall_block_kernel_phase3(int n, int k, int* graph, int start_x, int start_y) {
    // BlockDim is one dimensional (Straight along diagonal)
    // Blocks themselves are two dimensional
    const unsigned int i = start_y + blockIdx.y;
    const unsigned int j = start_x + blockIdx.x;
    const unsigned int bi = threadIdx.y;
    const unsigned int bj = threadIdx.x;

    __shared__ int A[BLOCK_DIM * BLOCK_DIM];
    __shared__ int B[BLOCK_DIM * BLOCK_DIM];
    __shared__ int C[BLOCK_DIM * BLOCK_DIM];

    C[bi*BLOCK_DIM + bj] = graph[(i*BLOCK_DIM + bi)*n + (j*BLOCK_DIM + bj)];
    // C[(bi + TH_DIM)*BLOCK_DIM + bj] = graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (j*BLOCK_DIM + bj)];
    // C[bi*BLOCK_DIM + (bj + TH_DIM)] = graph[(i*BLOCK_DIM + bi)*n + (j*BLOCK_DIM + (bj + TH_DIM))];
    // C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (j*BLOCK_DIM + (bj + TH_DIM))];

    A[bj*BLOCK_DIM + bi] = graph[(i*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + bj)];
    // A[bj*BLOCK_DIM + (bi + TH_DIM)] = graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + bj)];
    // A[(bj + TH_DIM)*BLOCK_DIM + bi] = graph[(i*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + (bj + TH_DIM))];
    // A[(bj + TH_DIM)*BLOCK_DIM + (bi + TH_DIM)] = graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + (bj + TH_DIM))];

    B[bi*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + bi)*n + (j*BLOCK_DIM + bj)];
    // B[(bi + TH_DIM)*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (j*BLOCK_DIM + bj)];
    // B[bi*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + bi)*n + (j*BLOCK_DIM + (bj + TH_DIM))];
    // B[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (j*BLOCK_DIM + (bj + TH_DIM))];

    __syncthreads();

    block_calc_rev_async(C, A, B, bi, bj);

    __syncthreads();

    graph[(i*BLOCK_DIM + bi)*n + (j*BLOCK_DIM + bj)] = C[bi*BLOCK_DIM + bj];
    // graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (j*BLOCK_DIM + bj)] = C[(bi + TH_DIM)*BLOCK_DIM + bj];
    // graph[(i*BLOCK_DIM + bi)*n + (j*BLOCK_DIM + (bj + TH_DIM))] = C[bi*BLOCK_DIM + (bj + TH_DIM)];
    // graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (j*BLOCK_DIM + (bj + TH_DIM))] = C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)];
}

void block_FW_cuda() {
    const int blocks = padding_n / BLOCK_DIM;
    const int row_b_size = BLOCK_DIM * padding_n;
    up_part_b_size = (blocks+1)/2;
    bottom_part_b_size = blocks/2;
    // printf("Up Blocks: %d, Bottom Blocks: %d\n", up_part_b_size, bottom_part_b_size);

    dim3 block_dim(TH_DIM, TH_DIM, 1);
    dim3 phase3_grid(blocks, blocks, 1);
    dim3 phase31_grid(blocks, up_part_b_size, 1);
    dim3 phase32_grid(blocks, bottom_part_b_size, 1);

    for (int k = 0; k < blocks; k++) {
        // Phase 1
        hipSetDevice(0);
        floyd_warshall_block_kernel_phase1<<<1, block_dim>>>(padding_n, k, Dist_cuda0);

        hipSetDevice(1);
        floyd_warshall_block_kernel_phase1<<<1, block_dim>>>(padding_n, k, Dist_cuda1);

        // Phase 2
        hipSetDevice(0);
        floyd_warshall_block_kernel_phase2<<<blocks, block_dim>>>(padding_n, k, Dist_cuda0);

        hipSetDevice(1);
        floyd_warshall_block_kernel_phase2<<<blocks, block_dim>>>(padding_n, k, Dist_cuda1);

        // Phase 3
        hipSetDevice(0);
        floyd_warshall_block_kernel_phase3<<<phase3_grid, block_dim>>>(padding_n, k, Dist_cuda0, 0, 0);
        // floyd_warshall_block_kernel_phase3<<<phase31_grid, block_dim>>>(padding_n, k, Dist_cuda0, 0, 0);

        // cudaSetDevice(1);
        // floyd_warshall_block_kernel_phase3<<<phase32_grid, block_dim>>>(padding_n, k, Dist_cuda1, 0, up_part_b_size);
    }
}

void block_FW_cuda0() {
    const int blocks = padding_n / BLOCK_DIM;
    const int row_b_size = BLOCK_DIM * padding_n;
    up_part_b_size = (blocks+1)/2;
    bottom_part_b_size = blocks/2;
    // printf("Up Blocks: %d, Bottom Blocks: %d\n", up_part_b_size, bottom_part_b_size);

    dim3 block_dim(TH_DIM, TH_DIM, 1);
    dim3 phase31_grid(blocks, up_part_b_size, 1);
    dim3 phase32_grid(blocks, bottom_part_b_size, 1);

    for (int k = 0; k < blocks; k++) {
        if(k < up_part_b_size){
            // Stage 1
            // printf("Round %d Before Copy\n", k);
            // printf("Matrix 0\n");
            // show_mat_cuda(Dist_cuda0, padding_n, 0);
            // printf("Matrix 1\n");
            // show_mat_cuda(Dist_cuda1, padding_n, 1);
            hipSetDevice(0);
            floyd_warshall_block_kernel_phase1<<<1, block_dim>>>(padding_n, k, Dist_cuda0);
            floyd_warshall_block_kernel_phase2<<<blocks, block_dim>>>(padding_n, k, Dist_cuda0);
            hipMemcpyPeer(&(Dist_cuda1[k * row_b_size]), 1, &(Dist_cuda0[k * row_b_size]), 0, SIZEOFINT * row_b_size); 
            // printf("%d. %d ~ %d Copy Done\n", k, k * row_b_size,  (k * row_b_size) + (row_b_size), row_b_size);

            hipDeviceSynchronize();

            hipSetDevice(1);
            // printf("After Copy\n");
            // printf("Matrix 0\n");
            // show_mat_cuda(Dist_cuda0, padding_n, 0);
            // printf("Matrix 1\n");
            // show_mat_cuda(Dist_cuda1, padding_n, 1);
            // printf("Down Part\n------------\n");
            floyd_warshall_block_kernel_phase2<<<blocks, block_dim>>>(padding_n, k, Dist_cuda1);
        }else{
            // Stage 2
            hipSetDevice(1);
            floyd_warshall_block_kernel_phase1<<<1, block_dim>>>(padding_n, k, Dist_cuda1);
            floyd_warshall_block_kernel_phase2<<<blocks, block_dim>>>(padding_n, k, Dist_cuda1);
            hipMemcpyPeer(&(Dist_cuda0[k * row_b_size]), 0, &(Dist_cuda1[k * row_b_size]), 1, SIZEOFINT * row_b_size); 
            // printf("%d. %d ~ %d Copy Done\n", k, k * row_b_size,  (k * row_b_size) + (row_b_size), row_b_size);

            hipDeviceSynchronize();

            hipSetDevice(0);
            // printf("Up Part\n------------\n");
            floyd_warshall_block_kernel_phase2<<<blocks, block_dim>>>(padding_n, k, Dist_cuda0);
        }

        // Phase 3
        hipSetDevice(0);
        floyd_warshall_block_kernel_phase3<<<phase31_grid, block_dim>>>(padding_n, k, Dist_cuda0, 0, 0);

        hipSetDevice(1);
        floyd_warshall_block_kernel_phase3<<<phase32_grid, block_dim>>>(padding_n, k, Dist_cuda1, 0, up_part_b_size);
    }
}

void block_FW_cuda1() {
    const int blocks = padding_n / BLOCK_DIM;
    const int row_b_size = BLOCK_DIM * padding_n;
    up_part_b_size = (blocks+1)/2;
    bottom_part_b_size = blocks/2;
    // printf("Up Blocks: %d, Bottom Blocks: %d\n", up_part_b_size, bottom_part_b_size);

    dim3 block_dim(TH_DIM, TH_DIM, 1);
    // dim3 phase3_grid(blocks, blocks, 1);
    dim3 phase31_grid(blocks, up_part_b_size, 1);
    dim3 phase32_grid(blocks, bottom_part_b_size, 1);

    const int num_stream = 2;
    hipStream_t streams[num_stream];
    for(int i=0; i<num_stream; i++) {hipStreamCreate(&streams[i]);}

    for (int k = 0; k < blocks; k++) {
        // Phase 1
        hipSetDevice(0);
        floyd_warshall_block_kernel_phase1<<<1, block_dim, 0>>>(padding_n, k, Dist_cuda0);

        hipSetDevice(1);
        floyd_warshall_block_kernel_phase1<<<1, block_dim, 0>>>(padding_n, k, Dist_cuda1);

        // Phase 2
        hipSetDevice(0);
        floyd_warshall_block_kernel_phase2<<<blocks, block_dim, 0>>>(padding_n, k, Dist_cuda0);

        hipSetDevice(1);
        floyd_warshall_block_kernel_phase2<<<blocks, block_dim, 0>>>(padding_n, k, Dist_cuda1);

        // Phase 3
        hipSetDevice(0);
        floyd_warshall_block_kernel_phase3<<<phase31_grid, block_dim, 0>>>(padding_n, k, Dist_cuda0, 0, 0);

        hipSetDevice(1);
        floyd_warshall_block_kernel_phase3<<<phase32_grid, block_dim, 0>>>(padding_n, k, Dist_cuda1, 0, up_part_b_size);

        // Transfer data to another GPU
        int next_k = k + 1;
        if(next_k < up_part_b_size){
            hipMemcpyPeerAsync(&(Dist_cuda1[next_k * row_b_size]), 1, &(Dist_cuda0[next_k * row_b_size]), 0, SIZEOFINT * row_b_size); 
        }else if(up_part_b_size < next_k < blocks){
            hipMemcpyPeerAsync(&(Dist_cuda0[next_k * row_b_size]), 0, &(Dist_cuda1[next_k * row_b_size]), 1, SIZEOFINT * row_b_size); 
        }
    }
    for(int i=0; i<num_stream; i++) {hipStreamDestroy(streams[i]);}
}

void block_FW_cuda2() {
    const int blocks = padding_n / BLOCK_DIM;
    const int row_b_size = BLOCK_DIM * padding_n;
    up_part_b_size = (blocks+1)/2;
    bottom_part_b_size = blocks/2;
    // printf("Up Blocks: %d, Bottom Blocks: %d\n", up_part_b_size, bottom_part_b_size);

    dim3 block_dim(TH_DIM, TH_DIM, 1);
    dim3 phase31_grid(blocks, up_part_b_size, 1);
    dim3 phase32_grid(blocks, bottom_part_b_size, 1);

    for (int k = 0; k < blocks; k++) {
        if(k < up_part_b_size){
            // Stage 1
            // printf("Round %d Before Copy\n", k);
            // printf("Matrix 0\n");
            // show_mat_cuda(Dist_cuda0, padding_n, 0);
            // printf("Matrix 1\n");
            // show_mat_cuda(Dist_cuda1, padding_n, 1);
            hipSetDevice(0);
            floyd_warshall_block_kernel_phase1<<<1, block_dim>>>(padding_n, k, Dist_cuda0);
            floyd_warshall_block_kernel_phase2<<<blocks, block_dim>>>(padding_n, k, Dist_cuda0);
            hipMemcpyPeer(&(Dist_cuda1[k * row_b_size]), 1, &(Dist_cuda0[k * row_b_size]), 0, SIZEOFINT * row_b_size); 
            // printf("%d. %d ~ %d Copy Done\n", k, k * row_b_size,  (k * row_b_size) + (row_b_size), row_b_size);

            hipDeviceSynchronize();

            hipSetDevice(1);
            // printf("After Copy\n");
            // printf("Matrix 0\n");
            // show_mat_cuda(Dist_cuda0, padding_n, 0);
            // printf("Matrix 1\n");
            // show_mat_cuda(Dist_cuda1, padding_n, 1);
            // printf("Down Part\n------------\n");
            floyd_warshall_block_kernel_phase2<<<blocks, block_dim>>>(padding_n, k, Dist_cuda1);
        }else{
            // Stage 2
            hipSetDevice(1);
            floyd_warshall_block_kernel_phase1<<<1, block_dim>>>(padding_n, k, Dist_cuda1);
            floyd_warshall_block_kernel_phase2<<<blocks, block_dim>>>(padding_n, k, Dist_cuda1);
            hipMemcpyPeer(&(Dist_cuda0[k * row_b_size]), 0, &(Dist_cuda1[k * row_b_size]), 1, SIZEOFINT * row_b_size); 
            // printf("%d. %d ~ %d Copy Done\n", k, k * row_b_size,  (k * row_b_size) + (row_b_size), row_b_size);

            hipDeviceSynchronize();

            hipSetDevice(0);
            // printf("Up Part\n------------\n");
            floyd_warshall_block_kernel_phase2<<<blocks, block_dim>>>(padding_n, k, Dist_cuda0);
        }

        // Phase 3
        hipSetDevice(0);
        floyd_warshall_block_kernel_phase3<<<phase31_grid, block_dim>>>(padding_n, k, Dist_cuda0, 0, 0);

        hipSetDevice(1);
        floyd_warshall_block_kernel_phase3<<<phase32_grid, block_dim>>>(padding_n, k, Dist_cuda1, 0, up_part_b_size);
    }
}

int main(int argc, char* argv[]) {
    input(argv[1]);
    // show_mat(getDistAddr(0, 0), padding_n);
    // printf("Original Matix\n");
    setup_DistCuda();
    // printf("Vertice: %d, Edge: %d, BLOCK_DIM: %d\n", n, m, BLOCK_DIM);
    block_FW_cuda1();
    back_DistCuda();
    // show_mat(getDistAddr(0, 0), n);
    
    output(argv[2]);
    // show_mat(Dist, padding_n);
    // printf("------------\n");
    // show_mat(Dist_s, n);
    return 0;
}