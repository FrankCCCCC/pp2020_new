#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define SIZEOFINT sizeof(int)
#define BLOCK_DIM 64
#define TH_DIM 32

const int INF = ((1 << 30) - 1);
int n, m, padding_n, pitch_k;
size_t pitch;
int *Dist, *Dist_s;
int *Dist_cuda;

void show_mat(int *start_p, int vertex_num){
    for(int i = 0; i < vertex_num; i++){
        for(int j = 0; j < vertex_num; j++){
            if(start_p[i * vertex_num + j] == INF){
                printf("INF\t  ");
            }else{
                printf("%d\t  ", start_p[i * vertex_num + j]);
            }   
        }
        printf("\n");
    }
}

void show_mat_cuda(int *start_p, int vertex_num, int padding_n, size_t pitch, int device_id){
    int *temp = (int*)malloc(SIZEOFINT * padding_n * padding_n);
    hipSetDevice(device_id);
    // cudaMemcpy(temp, start_p, (SIZEOFINT * vertex_num * vertex_num), cudaMemcpyDeviceToHost);
    hipMemcpy2D(temp, SIZEOFINT * padding_n, start_p, pitch, SIZEOFINT * padding_n, padding_n, hipMemcpyDeviceToHost);
    printf("---\n");
    for(int i = 0; i < vertex_num; i++){
        for(int j = 0; j < vertex_num; j++){
            if(temp[i * vertex_num + j] == INF){
                printf("INF\t  ");
            }else{
                printf("%d\t  ", temp[i * vertex_num + j]);
            }   
        }
        printf("\n");
    }
    printf("---\n");
}

void malloc_Dist(){
    hipHostAlloc(&Dist, SIZEOFINT * padding_n * padding_n, hipHostMallocPortable);
    // Dist = (int*)malloc(SIZEOFINT * padding_n * padding_n);
    Dist_s = (int*)malloc(SIZEOFINT * n * n);
}
int getDist(int i, int j){return Dist[i * padding_n + j];}
int *getDistAddr(int i, int j){return &(Dist[i * padding_n + j]);}
void setDist(int i, int j, int val){Dist[i * padding_n + j] = val;}

void setup_DistCuda(){
    // cudaMalloc((void **)&Dist_cuda, SIZEOFINT * padding_n * padding_n);
    // cudaMemcpy(Dist_cuda, Dist, (padding_n * padding_n * SIZEOFINT), cudaMemcpyHostToDevice);
    
    hipMallocPitch(&Dist_cuda, &pitch, SIZEOFINT * padding_n, padding_n);
    hipMemcpy2D(Dist_cuda, pitch, Dist, SIZEOFINT * padding_n, SIZEOFINT * padding_n, padding_n, hipMemcpyHostToDevice);
    pitch_k = ((int)pitch) / SIZEOFINT;
}
void back_DistCuda(){
    // cudaMemcpy(Dist, Dist_cuda, (padding_n * padding_n * SIZEOFINT), cudaMemcpyDeviceToHost);
    hipMemcpy2D(Dist, SIZEOFINT * padding_n, Dist_cuda, pitch, SIZEOFINT * padding_n, padding_n, hipMemcpyDeviceToHost);
}

void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);
    padding_n = ((n + BLOCK_DIM - 1) / BLOCK_DIM) * BLOCK_DIM;
    malloc_Dist();

    for (int i = 0; i < padding_n; i++) {
        for (int j = 0; j < padding_n; j++) {
            if (i == j) {
                setDist(i, j, 0);
                // Dist[i][j] = 0;
            } else {
                setDist(i, j, INF);
                // Dist[i][j] = INF;
            }
        }
    }

    int pair[3];
    for (int i = 0; i < m; i++) {
        fread(pair, sizeof(int), 3, file);
        setDist(pair[0], pair[1], pair[2]);
        // Dist[pair[0]][pair[1]] = pair[2];
    }
    // cudaMemcpy(Dist_cuda, Dist, (n * n * SIZEOFINT), cudaMemcpyHostToDevice);
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            // if (Dist[i][j] >= INF) Dist[i][j] = INF;
            if (getDist(i, j) >= INF) setDist(i, j, INF);
            Dist_s[i * n + j] = getDist(i, j);
        }
        // fwrite(Dist[i], sizeof(int), n, outfile);
        // fwrite(getDistAddr(i, 0), SIZEOFINT, n, outfile);
    }
    fwrite(Dist_s, sizeof(int), n * n, outfile);
    fclose(outfile);
}

__forceinline__
__device__ void block_calc(int* C, int* A, int* B, int bj, int bi) {
    for (int k = 0; k < BLOCK_DIM; k++) {
        int sum0 = A[bi*BLOCK_DIM + k] + B[k*BLOCK_DIM + bj];
        int sum1 = A[(bi + TH_DIM)*BLOCK_DIM + k] + B[k*BLOCK_DIM + bj];
        int sum2 = A[bi*BLOCK_DIM + k] + B[k*BLOCK_DIM + (bj + TH_DIM)];
        int sum3 = A[(bi + TH_DIM)*BLOCK_DIM + k] + B[k*BLOCK_DIM + (bj + TH_DIM)];

        C[bi*BLOCK_DIM + bj] = min(C[bi*BLOCK_DIM + bj], sum0);
        C[(bi + TH_DIM)*BLOCK_DIM + bj] = min(C[(bi + TH_DIM)*BLOCK_DIM + bj], sum1);
        C[bi*BLOCK_DIM + (bj + TH_DIM)] = min(C[bi*BLOCK_DIM + (bj + TH_DIM)], sum2);
        C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = min(C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)], sum3);
        __syncthreads();
    }
}

__forceinline__
__device__ void block_calc_rev_async(int* C, int* A, int* B, int bj, int bi) {
    #pragma unroll 10
    for (int k = 0; k < BLOCK_DIM; k++) {
        int sum0 = A[k*BLOCK_DIM + bi] + B[k*BLOCK_DIM + bj];
        int sum1 = A[k*BLOCK_DIM + (bi + TH_DIM)] + B[k*BLOCK_DIM + bj];
        int sum2 = A[k*BLOCK_DIM + bi] + B[k*BLOCK_DIM + (bj + TH_DIM)];
        int sum3 = A[k*BLOCK_DIM + (bi + TH_DIM)] + B[k*BLOCK_DIM + (bj + TH_DIM)];

        C[bi*BLOCK_DIM + bj] = min(C[bi*BLOCK_DIM + bj], sum0);
        C[(bi + TH_DIM)*BLOCK_DIM + bj] = min(C[(bi + TH_DIM)*BLOCK_DIM + bj], sum1);
        C[bi*BLOCK_DIM + (bj + TH_DIM)] = min(C[bi*BLOCK_DIM + (bj + TH_DIM)], sum2);
        C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = min(C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)], sum3);
    }
}

__global__ void floyd_warshall_block_kernel_phase1(int n, int k, int* graph) {
    const unsigned int bi = threadIdx.y;
    const unsigned int bj = threadIdx.x;

    __shared__ int C[BLOCK_DIM * BLOCK_DIM];

    // Transfer to temp shared arrays
    C[bi*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + bj)];
    C[(bi + TH_DIM)*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + bj)];
    C[bi*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + (bj + TH_DIM))];
    C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + (bj + TH_DIM))];

    __syncthreads();
    
    block_calc(C, C, C, bi, bj);

    // Transfer back to graph
    graph[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + bj)] = C[bi*BLOCK_DIM + bj];
    graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + bj)] = C[(bi + TH_DIM)*BLOCK_DIM + bj];
    graph[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + (bj + TH_DIM))] = C[bi*BLOCK_DIM + (bj + TH_DIM)];
    graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + (bj + TH_DIM))] = C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)];
}


__global__ void floyd_warshall_block_kernel_phase2(int n, int k, int* graph) {
    // BlockDim is one dimensional (Straight along diagonal)
    // Blocks themselves are two dimensional
    // Phase 2 1/2
    const unsigned int i = blockIdx.x;
    const unsigned int bi = threadIdx.y;
    const unsigned int bj = threadIdx.x;

    __shared__ int A[BLOCK_DIM * BLOCK_DIM];
    __shared__ int B[BLOCK_DIM * BLOCK_DIM];
    __shared__ int C[BLOCK_DIM * BLOCK_DIM];

    C[bi*BLOCK_DIM + bj] = graph[(i*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + bj)];
    C[(bi + TH_DIM)*BLOCK_DIM + bj] = graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + bj)];
    C[bi*BLOCK_DIM + (bj + TH_DIM)] = graph[(i*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + (bj + TH_DIM))];
    C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + (bj + TH_DIM))];

    B[bi*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + bj)];
    B[(bi + TH_DIM)*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + bj)];
    B[bi*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + (bj + TH_DIM))];
    B[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + (bj + TH_DIM))];

    __syncthreads();

    block_calc(C, C, B, bi, bj);

    graph[(i*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + bj)] = C[bi*BLOCK_DIM + bj];
    graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + bj)] = C[(bi + TH_DIM)*BLOCK_DIM + bj];
    graph[(i*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + (bj + TH_DIM))] = C[bi*BLOCK_DIM + (bj + TH_DIM)];
    graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + (bj + TH_DIM))] = C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)];

    // Phase 2 2/2

    C[bi*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + bi)*n + (i*BLOCK_DIM + bj)];
    C[(bi + TH_DIM)*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (i*BLOCK_DIM + bj)];
    C[bi*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + bi)*n + (i*BLOCK_DIM + (bj + TH_DIM))];
    C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (i*BLOCK_DIM + (bj + TH_DIM))];

    __syncthreads();

    block_calc(C, B, C, bi, bj);

    // Block C is the only one that could be changed
    graph[(k*BLOCK_DIM + bi)*n + (i*BLOCK_DIM + bj)] = C[bi*BLOCK_DIM + bj];
    graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (i*BLOCK_DIM + bj)] = C[(bi + TH_DIM)*BLOCK_DIM + bj];
    graph[(k*BLOCK_DIM + bi)*n + (i*BLOCK_DIM + (bj + TH_DIM))] = C[bi*BLOCK_DIM + (bj + TH_DIM)];
    graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (i*BLOCK_DIM + (bj + TH_DIM))] = C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)];
}


__global__ void floyd_warshall_block_kernel_phase3(int n, int k, int* graph, int start_x, int start_y) {
    // BlockDim is one dimensional (Straight along diagonal)
    // Blocks themselves are two dimensional
    const unsigned int j = start_x + blockIdx.x;
    const unsigned int i = start_y + blockIdx.y;
    const unsigned int bi = threadIdx.y;
    const unsigned int bj = threadIdx.x;

    __shared__ int A[BLOCK_DIM * BLOCK_DIM];
    __shared__ int B[BLOCK_DIM * BLOCK_DIM];
    __shared__ int C[BLOCK_DIM * BLOCK_DIM];

    C[bi*BLOCK_DIM + bj] = graph[(i*BLOCK_DIM + bi)*n + (j*BLOCK_DIM + bj)];
    C[(bi + TH_DIM)*BLOCK_DIM + bj] = graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (j*BLOCK_DIM + bj)];
    C[bi*BLOCK_DIM + (bj + TH_DIM)] = graph[(i*BLOCK_DIM + bi)*n + (j*BLOCK_DIM + (bj + TH_DIM))];
    C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (j*BLOCK_DIM + (bj + TH_DIM))];

    A[bj*BLOCK_DIM + bi] = graph[(i*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + bj)];
    A[bj*BLOCK_DIM + (bi + TH_DIM)] = graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + bj)];
    A[(bj + TH_DIM)*BLOCK_DIM + bi] = graph[(i*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + (bj + TH_DIM))];
    A[(bj + TH_DIM)*BLOCK_DIM + (bi + TH_DIM)] = graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + (bj + TH_DIM))];

    B[bi*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + bi)*n + (j*BLOCK_DIM + bj)];
    B[(bi + TH_DIM)*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (j*BLOCK_DIM + bj)];
    B[bi*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + bi)*n + (j*BLOCK_DIM + (bj + TH_DIM))];
    B[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (j*BLOCK_DIM + (bj + TH_DIM))];

    __syncthreads();

    block_calc_rev_async(C, A, B, bi, bj);

    __syncthreads();

    graph[(i*BLOCK_DIM + bi)*n + (j*BLOCK_DIM + bj)] = C[bi*BLOCK_DIM + bj];
    graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (j*BLOCK_DIM + bj)] = C[(bi + TH_DIM)*BLOCK_DIM + bj];
    graph[(i*BLOCK_DIM + bi)*n + (j*BLOCK_DIM + (bj + TH_DIM))] = C[bi*BLOCK_DIM + (bj + TH_DIM)];
    graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (j*BLOCK_DIM + (bj + TH_DIM))] = C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)];
}


void block_FW_cuda() {
    // int round = padding_n / B;
    const int blocks = padding_n / BLOCK_DIM;
    dim3 block_dim(TH_DIM, TH_DIM, 1);
    dim3 phase3_grid(blocks, blocks, 1);

    for (int k = 0; k < blocks; k++) {
        floyd_warshall_block_kernel_phase1<<<1, block_dim>>>(pitch_k, k, Dist_cuda);
        floyd_warshall_block_kernel_phase2<<<blocks, block_dim>>>(pitch_k, k, Dist_cuda);
        floyd_warshall_block_kernel_phase3<<<phase3_grid, block_dim>>>(pitch_k, k, Dist_cuda, 0, 0);
    }
}

int main(int argc, char* argv[]) {
    input(argv[1]);
    // show_mat(getDistAddr(0, 0), n);
    setup_DistCuda();
    // printf("Vertice: %d, Edge: %d, B: %d, Padding: %d\n", n, m, BLOCK_DIM, padding_n);
    block_FW_cuda();
    back_DistCuda();
    // show_mat(getDistAddr(0, 0), n);
    
    output(argv[2]);
    // show_mat(getDistAddr(0, 0), n);
    return 0;
}