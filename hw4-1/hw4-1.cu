#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define SIZEOFINT sizeof(int)
#define BLOCK_DIM 32
#define TH_DIM 32

const int INF = ((1 << 30) - 1);
const int blockdim_x = 32, blockdim_y = 32;
const dim3 block_dim(blockdim_x, blockdim_y);
const int B = 32;
const int Share_Mem_Size = 64;
const int Share_Mem_Size_sq = Share_Mem_Size * Share_Mem_Size;
const int Share_Mem_Row_Size = B;
int n, m, padding_n;
int *Dist, *Dist_s;
int *Dist_cuda;

// void show_mat(int *start_p, int vertex_num){
//     for(int i = 0; i < vertex_num; i++){
//         for(int j = 0; j < vertex_num; j++){
//             if(start_p[i * vertex_num + j] == INF){
//                 printf("INF\t  ");
//             }else{
//                 printf("%d\t  ", start_p[i * vertex_num + j]);
//             }   
//         }
//         printf("\n");
//     }
// }

void malloc_Dist(){
    // cudaHostAlloc(&Dist, SIZEOFINT * padding_n * padding_n, cudaHostAllocMapped);
    Dist = (int*)malloc(SIZEOFINT * padding_n * padding_n);
    Dist_s = (int*)malloc(SIZEOFINT * n * n);
}
int getDist(int i, int j){return Dist[i * padding_n + j];}
int *getDistAddr(int i, int j){return &(Dist[i * padding_n + j]);}
void setDist(int i, int j, int val){Dist[i * padding_n + j] = val;}

void setup_DistCuda(){
    hipMalloc((void **)&Dist_cuda, SIZEOFINT * padding_n * padding_n);
    hipMemcpy(Dist_cuda, Dist, (padding_n * padding_n * SIZEOFINT), hipMemcpyHostToDevice);
    // cudaHostGetDevicePointer(&Dist_cuda, Dist, 0);
}
void back_DistCuda(){
    hipMemcpy(Dist, Dist_cuda, (padding_n * padding_n * SIZEOFINT), hipMemcpyDeviceToHost);
}

void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);
    padding_n = ((n + B - 1) / B) * B;
    malloc_Dist();

    for (int i = 0; i < padding_n; i++) {
        for (int j = 0; j < padding_n; j++) {
            if (i == j) {
                setDist(i, j, 0);
                // Dist[i][j] = 0;
            } else {
                setDist(i, j, INF);
                // Dist[i][j] = INF;
            }
        }
    }

    int pair[3];
    for (int i = 0; i < m; i++) {
        fread(pair, sizeof(int), 3, file);
        setDist(pair[0], pair[1], pair[2]);
        // Dist[pair[0]][pair[1]] = pair[2];
    }
    // cudaMemcpy(Dist_cuda, Dist, (n * n * SIZEOFINT), cudaMemcpyHostToDevice);
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            // if (Dist[i][j] >= INF) Dist[i][j] = INF;
            if (getDist(i, j) >= INF) setDist(i, j, INF);
            Dist_s[i * n + j] = getDist(i, j);
        }
        // fwrite(Dist[i], sizeof(int), n, outfile);
        // fwrite(getDistAddr(i, 0), SIZEOFINT, n, outfile);
    }
    fwrite(Dist_s, sizeof(int), n * n, outfile);
    fclose(outfile);
}

__forceinline__
__device__ void block_calc(int* C, int* A, int* B, int bj, int bi) {
  for (int k = 0; k < BLOCK_DIM; k++) {
    int sum0 = A[bi*BLOCK_DIM + k] + B[k*BLOCK_DIM + bj];
    // int sum1 = A[(bi + TH_DIM)*BLOCK_DIM + k] + B[k*BLOCK_DIM + bj];
    // int sum2 = A[bi*BLOCK_DIM + k] + B[k*BLOCK_DIM + (bj + TH_DIM)];
    // int sum3 = A[(bi + TH_DIM)*BLOCK_DIM + k] + B[k*BLOCK_DIM + (bj + TH_DIM)];

    C[bi*BLOCK_DIM + bj] = min(C[bi*BLOCK_DIM + bj], sum0);
    // C[(bi + TH_DIM)*BLOCK_DIM + bj] = min(C[(bi + TH_DIM)*BLOCK_DIM + bj], sum1);
    // C[bi*BLOCK_DIM + (bj + TH_DIM)] = min(C[bi*BLOCK_DIM + (bj + TH_DIM)], sum2);
    // C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = min(C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)], sum3);
    __syncthreads();
  }
}

__forceinline__
__device__ void block_calc_rev_async(int* C, int* A, int* B, int bj, int bi) {
  #pragma unroll 5
  for (int k = 0; k < BLOCK_DIM; k++) {
    int sum0 = A[k*BLOCK_DIM + bi] + B[k*BLOCK_DIM + bj];
    // int sum1 = A[k*BLOCK_DIM + (bi + TH_DIM)] + B[k*BLOCK_DIM + bj];
    // int sum2 = A[k*BLOCK_DIM + bi] + B[k*BLOCK_DIM + (bj + TH_DIM)];
    // int sum3 = A[k*BLOCK_DIM + (bi + TH_DIM)] + B[k*BLOCK_DIM + (bj + TH_DIM)];

    C[bi*BLOCK_DIM + bj] = min(C[bi*BLOCK_DIM + bj], sum0);
    // C[(bi + TH_DIM)*BLOCK_DIM + bj] = min(C[(bi + TH_DIM)*BLOCK_DIM + bj], sum1);
    // C[bi*BLOCK_DIM + (bj + TH_DIM)] = min(C[bi*BLOCK_DIM + (bj + TH_DIM)], sum2);
    // C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = min(C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)], sum3);
  }
}

__global__ void floyd_warshall_block_kernel_phase1(int n, int k, int* graph) {
  const unsigned int bi = threadIdx.y;
  const unsigned int bj = threadIdx.x;

  __shared__ int C[BLOCK_DIM * BLOCK_DIM];

  // Transfer to temp shared arrays
  C[bi*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + bj)];
  // C[(bi + TH_DIM)*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + bj)];
  // C[bi*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + (bj + TH_DIM))];
  // C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + (bj + TH_DIM))];

  __syncthreads();
  
  block_calc(C, C, C, bi, bj);

  __syncthreads();

  // Transfer back to graph
  graph[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + bj)] = C[bi*BLOCK_DIM + bj];
  // graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + bj)] = C[(bi + TH_DIM)*BLOCK_DIM + bj];
  // graph[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + (bj + TH_DIM))] = C[bi*BLOCK_DIM + (bj + TH_DIM)];
  // graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + (bj + TH_DIM))] = C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)];
}


__global__ void floyd_warshall_block_kernel_phase2(int n, int k, int* graph) {
  // BlockDim is one dimensional (Straight along diagonal)
  // Blocks themselves are two dimensional
  const unsigned int i = blockIdx.x;
  const unsigned int bi = threadIdx.y;
  const unsigned int bj = threadIdx.x;

  __shared__ int A[BLOCK_DIM * BLOCK_DIM];
  __shared__ int B[BLOCK_DIM * BLOCK_DIM];
  __shared__ int C[BLOCK_DIM * BLOCK_DIM];

  C[bi*BLOCK_DIM + bj] = graph[(i*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + bj)];
  // C[(bi + TH_DIM)*BLOCK_DIM + bj] = graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + bj)];
  // C[bi*BLOCK_DIM + (bj + TH_DIM)] = graph[(i*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + (bj + TH_DIM))];
  // C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + (bj + TH_DIM))];

  B[bi*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + bj)];
  // B[(bi + TH_DIM)*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + bj)];
  // B[bi*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + (bj + TH_DIM))];
  // B[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + (bj + TH_DIM))];

  __syncthreads();

  block_calc(C, C, B, bi, bj);

  __syncthreads();

  graph[(i*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + bj)] = C[bi*BLOCK_DIM + bj];
  // graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + bj)] = C[(bi + TH_DIM)*BLOCK_DIM + bj];
  // graph[(i*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + (bj + TH_DIM))] = C[bi*BLOCK_DIM + (bj + TH_DIM)];
  // graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + (bj + TH_DIM))] = C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)];

  // Phase 2 1/2

  C[bi*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + bi)*n + (i*BLOCK_DIM + bj)];
  // C[(bi + TH_DIM)*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (i*BLOCK_DIM + bj)];
  // C[bi*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + bi)*n + (i*BLOCK_DIM + (bj + TH_DIM))];
  // C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (i*BLOCK_DIM + (bj + TH_DIM))];

  A[bi*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + bj)];
  // A[(bi + TH_DIM)*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + bj)];
  // A[bi*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + (bj + TH_DIM))];
  // A[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + (bj + TH_DIM))];

  __syncthreads();

  block_calc(C, A, C, bi, bj);

  __syncthreads();

  // Block C is the only one that could be changed
  graph[(k*BLOCK_DIM + bi)*n + (i*BLOCK_DIM + bj)] = C[bi*BLOCK_DIM + bj];
  // graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (i*BLOCK_DIM + bj)] = C[(bi + TH_DIM)*BLOCK_DIM + bj];
  // graph[(k*BLOCK_DIM + bi)*n + (i*BLOCK_DIM + (bj + TH_DIM))] = C[bi*BLOCK_DIM + (bj + TH_DIM)];
  // graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (i*BLOCK_DIM + (bj + TH_DIM))] = C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)];
}


__global__ void floyd_warshall_block_kernel_phase3(int n, int k, int* graph) {
  // BlockDim is one dimensional (Straight along diagonal)
  // Blocks themselves are two dimensional
  const unsigned int j = blockIdx.x;
  const unsigned int i = blockIdx.y;
  const unsigned int bi = threadIdx.y;
  const unsigned int bj = threadIdx.x;

  __shared__ int A[BLOCK_DIM * BLOCK_DIM];
  __shared__ int B[BLOCK_DIM * BLOCK_DIM];
  __shared__ int C[BLOCK_DIM * BLOCK_DIM];

  C[bi*BLOCK_DIM + bj] = graph[(i*BLOCK_DIM + bi)*n + (j*BLOCK_DIM + bj)];
  // C[(bi + TH_DIM)*BLOCK_DIM + bj] = graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (j*BLOCK_DIM + bj)];
  // C[bi*BLOCK_DIM + (bj + TH_DIM)] = graph[(i*BLOCK_DIM + bi)*n + (j*BLOCK_DIM + (bj + TH_DIM))];
  // C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (j*BLOCK_DIM + (bj + TH_DIM))];

  A[bj*BLOCK_DIM + bi] = graph[(i*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + bj)];
  // A[bj*BLOCK_DIM + (bi + TH_DIM)] = graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + bj)];
  // A[(bj + TH_DIM)*BLOCK_DIM + bi] = graph[(i*BLOCK_DIM + bi)*n + (k*BLOCK_DIM + (bj + TH_DIM))];
  // A[(bj + TH_DIM)*BLOCK_DIM + (bi + TH_DIM)] = graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (k*BLOCK_DIM + (bj + TH_DIM))];

  B[bi*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + bi)*n + (j*BLOCK_DIM + bj)];
  // B[(bi + TH_DIM)*BLOCK_DIM + bj] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (j*BLOCK_DIM + bj)];
  // B[bi*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + bi)*n + (j*BLOCK_DIM + (bj + TH_DIM))];
  // B[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)] = graph[(k*BLOCK_DIM + (bi + TH_DIM))*n + (j*BLOCK_DIM + (bj + TH_DIM))];

  __syncthreads();

  block_calc_rev_async(C, A, B, bi, bj);

  __syncthreads();

  graph[(i*BLOCK_DIM + bi)*n + (j*BLOCK_DIM + bj)] = C[bi*BLOCK_DIM + bj];
  // graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (j*BLOCK_DIM + bj)] = C[(bi + TH_DIM)*BLOCK_DIM + bj];
  // graph[(i*BLOCK_DIM + bi)*n + (j*BLOCK_DIM + (bj + TH_DIM))] = C[bi*BLOCK_DIM + (bj + TH_DIM)];
  // graph[(i*BLOCK_DIM + (bi + TH_DIM))*n + (j*BLOCK_DIM + (bj + TH_DIM))] = C[(bi + TH_DIM)*BLOCK_DIM + (bj + TH_DIM)];
}


void block_FW_cuda(int B) {
    // int round = padding_n / B;
    const int blocks = (padding_n + BLOCK_DIM - 1) / BLOCK_DIM;
    dim3 block_dim(TH_DIM, TH_DIM, 1);
    dim3 phase4_grid(blocks, blocks, 1);

  // std::cout << "Launching Kernels Blocks: " << blocks << " Size " << padding_n << "\n";
  for (int k = 0; k < blocks; k++) {
    floyd_warshall_block_kernel_phase1<<<1, block_dim>>>(padding_n, k, Dist_cuda);

    floyd_warshall_block_kernel_phase2<<<blocks, block_dim>>>(padding_n, k, Dist_cuda);

    floyd_warshall_block_kernel_phase3<<<phase4_grid, block_dim>>>(padding_n, k, Dist_cuda);
  }
}

int main(int argc, char* argv[]) {
    input(argv[1]);
    // show_mat(getDistAddr(0, 0), n);
    setup_DistCuda();
    // printf("Vertice: %d, Edge: %d, B: %d\n", n, m, B);
    block_FW_cuda(B);
    back_DistCuda();
    // show_mat(getDistAddr(0, 0), n);
    
    output(argv[2]);
    // show_mat(getDistAddr(0, 0), n);
    return 0;
}